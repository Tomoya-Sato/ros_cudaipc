#include "com_test/mod.h"
#include "com_test/gpuInfo.h"
#include <stdio.h>

#define DSIZE 256

inline void gassert(hipError_t err_code, const char *file, int line)
{
    if (err_code != hipSuccess) {
        fprintf(stderr, "Error: %s %s %d\n", hipGetErrorString(err_code), file, line);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

#define checkCudaErrors(val) gassert(val, __FILE__, __LINE__)

bool modifyDeviceMemory(com_test::gpuInfo::Request &req)
{

    hipIpcMemHandle_t my_handle;

    unsigned char buf[64];
    for (int i = 0; i < 64; i++) {
        buf[i] = req.devPtr[i];
    }

    memcpy((unsigned char*)&my_handle, buf, sizeof(my_handle));

    char *data;

    checkCudaErrors(hipIpcOpenMemHandle((void**)&data, my_handle, hipIpcMemLazyEnablePeerAccess));

    char *tmp = (char*)malloc(sizeof(char)*req.size);
    checkCudaErrors(hipMemcpy(tmp, data, sizeof(char)*req.size, hipMemcpyDeviceToHost));

    printf("gpu memory: %s\n", tmp);

    checkCudaErrors(hipIpcCloseMemHandle(data));
    free(tmp);

    return true;
}
