#include "com_test/mod.h"
#include "std_msgs/MultiArrayLayout.h"
#include "std_msgs/MultiArrayDimension.h"
#include "com_test/gpu_handle.h"
#include <stdio.h>

#define DSIZE 256

inline void gassert(hipError_t err_code, const char *file, int line)
{
    if (err_code != hipSuccess) {
        fprintf(stderr, "Error: %s %s %d\n", hipGetErrorString(err_code), file, line);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

#define checkCudaErrors(val) gassert(val, __FILE__, __LINE__)

void modifyDeviceMemory(com_test::gpu_handle msg)
{

    hipIpcMemHandle_t my_handle;

    unsigned char buf[64];
    for (int i = 0; i < 64; i++) {
        buf[i] = msg.data[i];
    }

    memcpy((unsigned char*)&my_handle, buf, sizeof(my_handle));

    char *data;

    checkCudaErrors(hipIpcOpenMemHandle((void**)&data, my_handle, hipIpcMemLazyEnablePeerAccess));

    char *tmp = (char*)malloc(sizeof(char)*(msg.size+1));
    checkCudaErrors(hipMemcpy(tmp, data, sizeof(char)*msg.size, hipMemcpyDeviceToHost));

    for (int i = 0; i < msg.size; i++) {
        printf("%c", tmp[i]);
    }
    printf("\n");

    checkCudaErrors(hipIpcCloseMemHandle(data));
    free(tmp);

    return;
}
