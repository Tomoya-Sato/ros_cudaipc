#include "com_test/mod.h"
#include "std_msgs/MultiArrayLayout.h"
#include "std_msgs/MultiArrayDimension.h"
#include "com_test/gpu_handle.h"
#include <stdio.h>

#define DSIZE 256

inline void gassert(hipError_t err_code, const char *file, int line)
{
    if (err_code != hipSuccess) {
        fprintf(stderr, "Error: %s %s %d\n", hipGetErrorString(err_code), file, line);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

#define checkCudaErrors(val) gassert(val, __FILE__, __LINE__)

void modifyDeviceMemory(com_test::gpu_handle msg)
{

    hipIpcMemHandle_t my_handle;

    unsigned char buf[64];
    for (int i = 0; i < 64; i++) {
        buf[i] = msg.data[i];
    }

    memcpy((unsigned char*)&my_handle, buf, sizeof(my_handle));

    char *data;

    checkCudaErrors(hipIpcOpenMemHandle((void**)&data, my_handle, hipIpcMemLazyEnablePeerAccess));

    char tmp[DSIZE];
    checkCudaErrors(hipMemcpy(tmp, data, sizeof(char)*DSIZE, hipMemcpyDeviceToHost));

    printf("%s\n", tmp);

    return;
}
