#include "hip/hip_runtime.h"
#include "com_test/hip/hip_runtime.h"
#include <stdio.h>
#include <sstream>
#include <string.h>

#include "com_test/common.h"

#define DSIZE 256
inline void gassert(hipError_t err_code, const char *file, int line)
{
    if (err_code != hipSuccess) {
        fprintf(stderr, "Error: %s %s %d\n", hipGetErrorString(err_code), file, line);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

#define checkCudaErrors(val) gassert(val, __FILE__, __LINE__)

GpuIpcTest::GpuIpcTest()
{
    return;
}

extern "C" __global__ void gpuMemorySet(int *data, int num)
{
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = id; i < num; i+=stride) {
        data[i] = i;
    }
}

unsigned char* GpuIpcTest::get_handle_buffer()
{
    hipIpcMemHandle_t my_handle;
    handle_buffer = (unsigned char*)malloc(sizeof(my_handle)+1);

    char str[DSIZE];
    scanf("%s", str);

    checkCudaErrors(hipMalloc((void**)&data, DSIZE*sizeof(char)));
    checkCudaErrors(hipMemcpy(data, str, DSIZE*sizeof(char), hipMemcpyHostToDevice));

    char tmp[DSIZE];
    checkCudaErrors(hipMemcpy(tmp, data, DSIZE*sizeof(char), hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipIpcGetMemHandle(&my_handle, data));

    unsigned char *buf = (unsigned char*)malloc(sizeof(unsigned char)*(sizeof(my_handle)+1));
    memset(handle_buffer, 0, sizeof(my_handle)+1);
    memcpy(handle_buffer, (unsigned char*)(&my_handle), sizeof(my_handle));

    memcpy(buf, handle_buffer, sizeof(my_handle));

    return buf;
}

void GpuIpcTest::printDeviceMemory()
{
    char tmp[DSIZE];
    checkCudaErrors(hipMemcpy(tmp, data, sizeof(char)*DSIZE, hipMemcpyDeviceToHost));

    printf("%s\n", tmp);
}

void GpuIpcTest::free_resources()
{
    checkCudaErrors(hipFree(data));
    free(handle_buffer);
    
    return;
}
