#include "hip/hip_runtime.h"
#include "com_test/hip/hip_runtime.h"
#include <stdio.h>
#include <sstream>
#include <string.h>

#include "com_test/common.h"

#define DSIZE 256
inline void gassert(hipError_t err_code, const char *file, int line)
{
    if (err_code != hipSuccess) {
        fprintf(stderr, "Error: %s %s %d\n", hipGetErrorString(err_code), file, line);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

#define checkCudaErrors(val) gassert(val, __FILE__, __LINE__)

GpuIpcTest::GpuIpcTest()
{
    bufCounter = 0;
    return;
}

extern "C" __global__ void gpuMemorySet(int *data, int num)
{
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = id; i < num; i+=stride) {
        data[i] = i;
    }
}

void GpuIpcTest::initGpuMemory()
{
    hipMalloc((void**)&data, DSIZE*sizeof(char));

    return;
}

unsigned char* GpuIpcTest::getHandleBuffer()
{
    hipIpcMemHandle_t my_handle;
    handle_buffer = (unsigned char*)malloc(sizeof(my_handle)+1);

    checkCudaErrors(hipMemcpy(data, buf[0], sizeBuffer[0]*sizeof(char), hipMemcpyHostToDevice));

    checkCudaErrors(hipIpcGetMemHandle(&my_handle, data));

    memset(handle_buffer, 0, sizeof(my_handle)+1);
    memcpy(handle_buffer, (unsigned char*)(&my_handle), sizeof(my_handle));

    free(buf[0]);
    for (int i = 0; i < bufCounter; i++) {
        buf[i] = buf[i+1];
        sizeBuffer[i] = sizeBuffer[i+1];
    }

    bufCounter--;

    return handle_buffer;
}

void GpuIpcTest::storeBuffer(const char *str, const int size)
{
    if (bufCounter == 10) {
        fprintf(stderr, "pointcloud buffer overflow.\n");
        exit(EXIT_FAILURE);
    }
    buf[bufCounter] = (char*)malloc(sizeof(char)*size);
    memcpy(buf[bufCounter], str, sizeof(char)*size);

    sizeBuffer[bufCounter] = size;

    bufCounter++;
    
    return;
}

int GpuIpcTest::getSize() {
    return sizeBuffer[0];
}

void GpuIpcTest::freeHandleBuffer()
{
    free(handle_buffer);

    return;
}

void GpuIpcTest::printDeviceMemory()
{
    char tmp[DSIZE];
    checkCudaErrors(hipMemcpy(tmp, data, sizeof(char)*DSIZE, hipMemcpyDeviceToHost));

    printf("%s\n", tmp);
}

void GpuIpcTest::freeResources()
{
    checkCudaErrors(hipFree(data));
    
    return;
}

bool GpuIpcTest::notEmpty()
{
    if (bufCounter == 0) return false;
    else return true;
}
