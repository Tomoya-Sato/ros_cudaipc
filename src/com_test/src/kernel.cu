#include "hip/hip_runtime.h"
#include "com_test/hip/hip_runtime.h"
#include <stdio.h>
#include <sstream>
#include <string.h>

#include "com_test/common.h"

#define DSIZE 256
inline void gassert(hipError_t err_code, const char *file, int line)
{
    if (err_code != hipSuccess) {
        fprintf(stderr, "Error: %s %s %d\n", hipGetErrorString(err_code), file, line);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

#define checkCudaErrors(val) gassert(val, __FILE__, __LINE__)

GpuIpcTest::GpuIpcTest()
{
    return;
}

extern "C" __global__ void gpuMemorySet(int *data, int num)
{
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = id; i < num; i+=stride) {
        data[i] = i;
    }
}

void GpuIpcTest::initGpuMemory()
{
    hipMalloc((void**)&data, DSIZE*sizeof(char));

    return;
}

unsigned char* GpuIpcTest::getHandleBuffer()
{
    hipIpcMemHandle_t my_handle;
    handle_buffer = (unsigned char*)malloc(sizeof(my_handle)+1);

    char str[DSIZE];
    scanf("%s", str);

    checkCudaErrors(hipMemcpy(data, str, DSIZE*sizeof(char), hipMemcpyHostToDevice));

    char tmp[DSIZE];
    checkCudaErrors(hipMemcpy(tmp, data, DSIZE*sizeof(char), hipMemcpyDeviceToHost));
    
    checkCudaErrors(hipIpcGetMemHandle(&my_handle, data));

    memset(handle_buffer, 0, sizeof(my_handle)+1);
    memcpy(handle_buffer, (unsigned char*)(&my_handle), sizeof(my_handle));

    return handle_buffer;
}

void GpuIpcTest::freeHandleBuffer()
{
    free(handle_buffer);

    return;
}

void GpuIpcTest::printDeviceMemory()
{
    char tmp[DSIZE];
    checkCudaErrors(hipMemcpy(tmp, data, sizeof(char)*DSIZE, hipMemcpyDeviceToHost));

    printf("%s\n", tmp);
}

void GpuIpcTest::freeResources()
{
    checkCudaErrors(hipFree(data));
    free(handle_buffer);
    
    return;
}
